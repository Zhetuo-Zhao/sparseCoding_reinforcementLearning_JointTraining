#include "hip/hip_runtime.h"
#include "ImageLoader.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>



 __global__ void real2complex (float *a, hipfftComplex *c, int N) 
{ 
	
	int idx = blockIdx.x*blockDim.x+threadIdx.x; 
	int idy = blockIdx.y*blockDim.y+threadIdx.y; 
	if ( idx < N && idy <N) 
	{ 
	 int index = idx + idy*N; 
	 c[index].x = a[index]; 
	 c[index].y = 0.f; 
	} 
	__syncthreads();
}

 /*compute idx and idy, the location of the element in the original NxN array*/ 
 __global__ void complex2real_scaled (hipfftComplex *c, float *a, int M, int N, float scale) 
{ 

	int idx = blockIdx.x*blockDim.x+threadIdx.x; 
	int idy = blockIdx.y*blockDim.y+threadIdx.y; 
	if ( (idx>=(M-N)/2)&&(idx<M-(M-N)/2) && (idy>=(M-N)/2)&&(idy<M-(M-N)/2)) 
	 { 
		int index = idx + idy*M; 
		int index2 = idx-(M-N)/2 + (idy-(M-N)/2)*N; 
		a[index2] = scale*c[index].x ; 
	 }
	__syncthreads();
} 

inline __device__ void mulAndScale(hipComplex& a, const hipComplex& b, const float& c){
     hipComplex t = {c * (a.x * b.x - a.y * b.y), c * (a.y * b.x + a.x * b.y)};
     a = t;
 }

 __global__ void modulateAndNormalize_kernel(hipComplex *d_Dst, hipComplex *d_Src, int N, float c )
 {
    int idx = blockIdx.x*blockDim.x+threadIdx.x; 
	int idy = blockIdx.y*blockDim.y+threadIdx.y; 
	hipComplex a,b;
	
	if ( idx < N && idy <N) 
	{ 
	 int i = idx + idy*N; 
		 a = d_Src[i];
		 b = d_Dst[i];
		 mulAndScale(a, b, c);
		 d_Dst[i] = a;
	}
	__syncthreads();
 }

void ImageLoader::imgWhitening()
{

	real2complex<<< dim3(16,16), dim3(16,16)>>>(leftWindow, d_dataL, FILT_WIDTH);
	real2complex<<< dim3(16,16), dim3(16,16)>>>(rightWindow, d_dataR, FILT_WIDTH);
	hipDeviceSynchronize();

	hipfftExecC2C(fftPlan, (hipfftComplex *)d_dataL, (hipfftComplex *)d_DataSpectrumL,HIPFFT_FORWARD );
	hipfftExecC2C(fftPlan, (hipfftComplex *)d_dataR, (hipfftComplex *)d_DataSpectrumR,HIPFFT_FORWARD );


	modulateAndNormalize_kernel<<<dim3(16,16), dim3(16,16)>>> (d_DataSpectrumL, d_filter, FILT_WIDTH , 1);
	modulateAndNormalize_kernel<<<dim3(16,16), dim3(16,16)>>> (d_DataSpectrumR, d_filter, FILT_WIDTH , 1);
	hipDeviceSynchronize();


	hipfftExecC2C(fftPlan, (hipfftComplex *)d_DataSpectrumL, (hipfftComplex *)d_resultL,HIPFFT_BACKWARD );
	hipfftExecC2C(fftPlan, (hipfftComplex *)d_DataSpectrumR, (hipfftComplex *)d_resultR,HIPFFT_BACKWARD );

	complex2real_scaled<<< dim3(16,16), dim3(16,16)>>>(d_resultL, leftWindowWh, FILT_WIDTH, FILT_WIDTH, float(1)/(FILT_WIDTH*FILT_WIDTH) );
	complex2real_scaled<<< dim3(16,16), dim3(16,16)>>>(d_resultR, rightWindowWh, FILT_WIDTH, FILT_WIDTH, float(1)/(FILT_WIDTH*FILT_WIDTH));


}

void ImageLoader::initFilt(float* f)
{
	filtdata=f;
	hipMemcpy(d_filterReal,filtdata,sizeof(float)*FILT_WIDTH*FILT_WIDTH,hipMemcpyHostToDevice);
	real2complex<<< dim3(16,16), dim3(16,16)>>>(d_filterReal, d_filter, FILT_WIDTH);
}